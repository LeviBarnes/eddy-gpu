#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math_functions.h>
#include <math.h>
#include <string.h>
#include <cstdlib>
#include <sys/timeb.h>
#include <assert.h>
#include <time.h>
#include <vector>
#include <boost/math/special_functions/beta.hpp>
#include <boost/math/special_functions/gamma.hpp>
#include "definitions.cuh"
//#define MAX_THREADS 1024
//#define MAX_THREADS 23
//#define MAX_GENES 50
//#ifdef _WIN32
//const char DIR[20] = "PRIORS\\";
//#endif
//#ifdef linux
//const char DIR[20] = "PRIORS/";
//#include <errno.h>
//#endif
//extern "C"
//{
//#include "incomplete_beta_function.h"
//#include "beta_function.h"
//}
//#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
//void printVec(int** a, int n);

static void HandleError(hipError_t err, const char *file, int line)
{       
        if (err != hipSuccess)
        {       
                printf("%s in %s at line %d\n", hipGetErrorString(err),
                        file, line);
                exit(EXIT_FAILURE);
        }
}


int main(int argc, char *argv[])
{
	//looking at GPU properties
	int nDevices;
	//int maxBlocks;
	int maxThreads;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Processor Clock Rate (KHz): %d\n", prop.clockRate);
		printf("  Device Max Number of Blocks: %d\n",
			prop.maxGridSize[1]);
		
		printf("  Device Max Number of Threads per Block: %d\n",
			prop.maxThreadsPerBlock);
		maxThreads = prop.maxThreadsPerBlock;
		printf("  Device Max Number of Compute Indices: %d\n",
			prop.maxGridSize[1] * prop.maxThreadsPerBlock);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		printf("  Compute Capability : %d.%d\n", prop.major, prop.minor);
		printf("  Device has %d SMs\n", prop.multiProcessorCount);
		printf("  This device can run multiple kernels simultaneously : %d \n\n",
			prop.concurrentKernels);
		

	}

	int startT = getMilliCount();
	int start1 = getMilliCount();
	//data grab routine *************************************************************************************
	//command line arguments parser--------------------------------------------------------------------------

	char *inputFile = NULL;
	char *classFile = NULL;
	char *genesetFile = NULL;
	//limits the number of parents a node can have
	int parentCap = 0;
	//number of permutations
	int perms = 0; // 1 permutation
	double pw = 1.0; //default - no prior weight
	double pThreshold = .05; //default value
    double theta = 0.8;
    double lambda = 2.0;
    double thresh;

	//-d for input
	//-g for geneset
	//-c for class
	//-mp for max parents
	//-p for p threshold value
	//-r number of permutations
	//-t for theta 
	//-l for lambda 
	//-pw prior weight = [0,1]
	//loop through argv, determining location of each arg parameter
	for (int i = 1; i < argc; i++)
	{
		if (strcmp(argv[i], "-d") == 0)
			inputFile = argv[i + 1];
		else if (strcmp(argv[i], "-g") == 0)
			genesetFile = argv[i + 1];
		else if (strcmp(argv[i], "-c") == 0)
			classFile = argv[i + 1];
		else if (strcmp(argv[i], "-l") == 0)
			lambda = atof(argv[i + 1]);
		else if (strcmp(argv[i], "-t") == 0)
			theta = atof(argv[i + 1]);
		else if (strcmp(argv[i], "-mp") == 0)
			parentCap = atoi(argv[i + 1]);
		else if (strcmp(argv[i], "-help") == 0 || strcmp(argv[i], "--help") == 0)
			printf("Required arguments : \n -d input.txt\n -c classfile.txt\n -g geneset.txt -mp # of parents\n -p pvalue for independence testing\n");
		else if (strcmp(argv[i], "-r") == 0)
			perms = atoi(argv[i + 1]);
		else if (strcmp(argv[i], "-pw") == 0)
			pw = atof(argv[i + 1]);
		else if(strcmp(argv[i], "-p") == 0)
			pThreshold = atof(argv[i+1]);
	}

	
	
	
	
	//set to defaults if no arguments are included
	if (inputFile == NULL)	{ printf("Invalid input file entered. Exiting...\n"); exit(EXIT_FAILURE); }
	if (classFile == NULL)	{ printf("Invalid class file entered. Exiting...\n"); exit(EXIT_FAILURE); }
	if (genesetFile == NULL)	{ printf("Invalid geneset file entered. Exiting...\n"); exit(EXIT_FAILURE); }
	printf("%s\n", genesetFile);
	if(pThreshold < 0.0 || pThreshold > 1.0)
	{
		pThreshold = .05; //default if out of range
	}
	printf("p Threshold  = %f\n", pThreshold); 
	//set maxparents to default 3 if not set in command line arguments
	if (parentCap <= 0)
	{
		//should normally be run with 3 which will make it for a total of 4
		parentCap = 3;
	}
	printf("Max parents = %d\n", parentCap);
	const int MAX_PARENTS = parentCap;
	if (perms <= 0)
	{
		perms = 100; //default
	}
	printf("Permutations = %d\n", perms);
	if (pw < 0.0 || pw > 1.0)
	{
		pw = 1.0; //no prior knowledge
	}
	printf("pw : %f\n", pw);

    thresh = pow(theta,(1/lambda));
	printf("lambda : %f, theta : %f, threshold : %f\n", lambda, theta, thresh);
	
	//
	//SK: to use the current codes in kernel1.cu as is
	//    if pw == 1.0 means no prior knowledge as described above
	//
	//    However, pw in command line option should such that 
	//       pw = 0 means no prior knowledge) and 
	//       pw = 1 means full prior knowledge
	//    to be consistent with the manuscript's description of
	//       prior knowledge and Eq. 2 in https://www.ncbi.nlm.nih.gov/pmc/articles/PMC4721243/
	//
	pw = pow(pw, (1/lambda));
	
	//end command line parser---------------------------------------------------------------------------------------

	//expression data
	FILE *fp = fopen(inputFile, "r");
	
	//class 
	FILE *fp2 = fopen(classFile, "r");

	//FILE * fp3 = fopen("geneset10.txt", "r");
	FILE *fp3 = fopen(genesetFile, "r");

	FILE *results = fopen("results.txt", "w");
	

	//check that files are working
	if (fp == NULL)	{ printf("Expression file is NULL. error number is : %d\n", strerror(errno)); exit(EXIT_FAILURE); }
	if (fp2 == NULL){ printf("Class File File is NULL. error number is : %d\n", strerror(errno)); exit(EXIT_FAILURE); }
	if (fp3 == NULL){ printf("Gene List File is NULL. error number is : %d\n", strerror(errno)); exit(EXIT_FAILURE); }
	if (results == NULL){ printf("results file is NULL. error number is : %d\n", strerror(errno)); exit(EXIT_FAILURE); }
	

	//allocate memory for file reads
	char buf[200000];
	int numsamples = 0;
	int numgenes = 0, genesetlength = 0;
	int numclass1, numclass2;
	char sampnames[5000][50];
	char classnames[5000][50];
	char genesetgenes[100][40];
	int classids[5000];
	char class1[50], class2[50];
	char genenames[20500][16]; //used to be [20000][10]- not big enough --> changed stack size to 1.5MB
	int genesetindexintodata[200]; //updated size from 50 to accomodate more genes
	//int classindexintodata[100][2];
	int i, j;// , index;
	//int jindex1, jindex2;
	int *data;
	int *transferdata1;
	int *transferdata2;
	char *token;


	//loads expression file into buffer
	fgets(buf, sizeof(buf), fp);
	token = strtok(buf, "\t");
	token = strtok(NULL, "\t");

	// Skip first word "Genelist"
	//load  samplenames from buffer and count number of samples
	while (token != NULL) {
		strcpy(sampnames[numsamples], token);
		numsamples++;
		token = strtok(NULL, "\t");
	}

	for (int i = 0; i < numsamples; i++) {
		// Get rid of extra empty "sample" caused by trailing tab
		if (strlen(sampnames[i]) == 1) numsamples--;
	}

	printf("%d samples\n", numsamples);

	while (fgets(buf, sizeof(buf), fp)) {
		token = strtok(buf, "\t");
		strcpy(genenames[numgenes], token);
		//    printf("%s\n", genenames[numgenes]);
		numgenes++;
	}

	numgenes--;
	printf("%d genes\n", numgenes);

	data = (int *)malloc(numgenes*numsamples*sizeof(int));
	//reset file position to 0
	fseek(fp, 0, 0);
	// Skip first line
	fgets(buf, sizeof(buf), fp);
	for (i = 0; i < numgenes; i++) {
		fgets(buf, sizeof(buf), fp);
		token = strtok(buf, "\t");
		for (j = 0; j < numsamples; j++) {
			token = strtok(NULL, "\t");
			assert(i * numsamples + j < numgenes * numsamples);
			sscanf(token, "%d", &data[i*numsamples + j]);
		}
	}

	fclose(fp);
	fgets(buf, sizeof(buf), fp2);
	token = strtok(buf, "\t");
	for (i = 0; i < numsamples; i++) {
		strcpy(classnames[i], token);
		token = strtok(NULL, "\t");
	}
	for (i = 1; i < numsamples; i++) {
		if (strcmp(classnames[i], classnames[0])) break;
	}
	strcpy(class1, classnames[0]);
	strcpy(class2, classnames[i]);
	numclass1 = 0;
	numclass2 = 0;

	//check if classfile had newline character at end of final classname, preventing 
	//strcmp from working
	if (classnames[numsamples - 1][strlen(classnames[numsamples - 1]) - 1] == '\n')
	{
		classnames[numsamples - 1][strlen(classnames[numsamples - 1]) - 1] = '\0';
	}

	for (i = 0; i < numsamples; i++) {
		if (!strcmp(classnames[i], class1)) {
			numclass1++;
			classids[i] = 0;

		}
		if (!strcmp(classnames[i], class2)) {
			numclass2++;
			classids[i] = 1;
		}

	}

	//should this be done? when reading in CTRP sample data not all classids are filled
	//this leads to a problem when reshuffling them later - only finding 201/202 supposed samples
	/*int revisedSamples = 0;
	for (int i = 0; i < numsamples; i++)
	{
		if (classids[i] == 1 || classids[i] == 0)
		{
			revisedSamples++;
		}
	}
	printf("revised : %d original : %d\n", revisedSamples, numsamples);
	numsamples = revisedSamples;*/

	//  printf("\n");
	printf("Classes: %d %s, %d %s\n", numclass1, class1, numclass2, class2);
	fclose(fp2);
	clock_t cpuTime = clock(), diff;
	//-----------------------------------------------------------------------------------
	//-----------------------------------------------------------------------------------

	fprintf(results, "%s\t %s\t %s\t\n", class2, "JS", "P");
	while (fgets(buf, sizeof(buf), fp3))
	{
		//check if beginning of file is newline
		if (buf[0] == '\n')
		{
			continue;
		}
		//ensures that classids are restored to a preshuffled state
		//for the first run of each pathway before permutations begin
		for (int k = 0; k < numsamples; k++)
		{
			if (strcmp(classnames[k], class1) == 0)
			{
				classids[k] = 0;
			}
			if (strcmp(classnames[k], class2) == 0)
			{
				classids[k] = 1;
			}

		}

		char *pathwayName = strtok(buf, "\n");
		//fgets(buf, sizeof(buf), fp3);
		//token = strtok(buf, "\t");
		token = strtok(pathwayName, "\t");
		token = strtok(NULL, "\t");
		// Get first word "Geneset"
		token = strtok(NULL, "\t");
		// Skip second word "URL"
		genesetlength = 0;
		while (token != NULL) {
			strcpy(genesetgenes[genesetlength], token);
			genesetlength++;
			token = strtok(NULL, "\t");
		}

		printf("%s\n", pathwayName);
		
		//-------------------------------------------------------------------------
		// Get rid of trailing carriage return on last gene name
		//no longer needed because strok with pathwayName eliminates newline character
		//genesetgenes[genesetlength - 1][strlen(genesetgenes[genesetlength - 1]) - 1] = '\0';

		printf("%d genes in geneset\n", genesetlength);
		//fclose(fp3);

		//should this be moved to after adjusting genesetlength?
		//transferdata1 = (int *)malloc(genesetlength*numclass1*sizeof(int));
		//transferdata2 = (int *)malloc(genesetlength*numclass2*sizeof(int));
		
		//accounts for any missing/extra genes
		int indexPos = 0;
		for (i = 0; i < genesetlength; i++) {
			int flagFound = 0;
			genesetindexintodata[i] = -1;
			for (j = 0; j < numgenes; j++) {
				if (!strcmp(genenames[j], genesetgenes[i])) {
					flagFound = 1;
					//fill genesetgenes only with genes that are being evaluated
					strcpy(genesetgenes[indexPos], genenames[j]);
					genesetindexintodata[indexPos] = j;
					break;
				}
			}
			//	printf ("Gene %d index: %d %d %s %s\n",i,genesetindexintodata[i],j,genenames[genesetindexintodata[i]],genesetgenes[i]);
			if (flagFound)
			{
				indexPos++;
			}
		}
		transferdata1 = (int *)malloc(genesetlength*numclass1*sizeof(int));
		transferdata2 = (int *)malloc(genesetlength*numclass2*sizeof(int));
		//delete genes that shouldn't be in gene list
		for (int k = indexPos; k < genesetlength; k++)	{ genesetgenes[k][0] = '\0'; }
		//adjust # of genes
		genesetlength = indexPos;
		printf("Adjusted genes : %d\n", genesetlength);
		
		
		//prior knowledge load data into binary matrix-------------------------------------------------------------
		
		int *priorMatrix = (int *)calloc(genesetlength * genesetlength, sizeof(int)); //array to hold prior knowledge matrix

		//look into \PRIORS folder
		char directory[300];; //directory for prior files
		strcpy(directory, DIR); //load folder path depending on if unix or windows
		char fileName[1000]; //name of prior file
		strcpy(fileName, pathwayName);
		strcat(fileName, ".prior"); //take pathwayname and add .prior to get file path
		strcat(directory, fileName);
		if(strstr(directory, "\r") != NULL){
			printf("File problem! Uses window endings!\n");
		}
		
		FILE *priorFile = fopen(directory, "r"); //open prior knowledge file
		printf("file : %s\n", directory);
		int priorFlag = 1; //1 = files found 0 = no file found
		if (priorFile == NULL)
		{
			printf("No prior file exists. Computing without prior knowledge\n");
			priorFlag = 0;
		}
		//fill prior Matrix
		char priorBuffer[100];
		while (priorFlag && fgets(priorBuffer, sizeof(priorBuffer), priorFile))
		{
			char *tok = strtok(priorBuffer, "\t");
			//printf("gene1 : %s\n", tok);
			int insideFlag = 0;
			int row = -1, col = -1;
			for (int k = 0; k < genesetlength; k++)
			{
				if (strcmp(genesetgenes[k], tok) == 0)
				{
					insideFlag = 1;
					row = k;
					break;
				}
			}

			if (insideFlag == 0)
				continue;
			tok = strtok(NULL, "\t");
			//printf("relationship : %s\n", tok);
			if (strcmp(tok, "neighbor-of") == 0)
			{
				continue;
			}
			tok = strtok(NULL, "\t");
			tok[strlen(tok) - 1] = '\0';
			//printf("gene2 : %s\n", tok);
			insideFlag = 0;
			for (int k = 0; k < genesetlength; k++)
			{
				if (strcmp(genesetgenes[k], tok) == 0)
				{
					insideFlag = 1;
					col = k;
					break;
				}
			}
			if (insideFlag == 0)
				continue;
			assert(row > -1 && col > -1 && row < genesetlength && col < genesetlength);
			*(priorMatrix + row * genesetlength + col) = 1;
			*(priorMatrix + col * genesetlength + row) = 1;
		}

		if (priorFlag == 1) //only try closing file if it was open to begin with
		{
			fclose(priorFile);
		}
			
		//begin permutation loop
		int n;
		int x;

		//stores js values across permutations for p value calcs
		double *jsVals = (double *)malloc(sizeof(double) * perms);
		//int *triAry2;
		//int *triAry3;
		//used to print network/bdeu score files
		int first_unisum;
		int first_scaler;
		int *first_uniNodes;
		int *first_uniEdges;
		int *first_uniEpn;
		double *first_lval1 = NULL;
		int first_numEdges;
		int *uniqueNetIds = NULL;
		//used to calculate edgeList without parent limit after permuatations finished - stores 1st permutation data
		int *edgeListData1 = NULL;
		int *edgeListData2 = NULL;
		//int *initialSpacr = NULL;
		//int *initialFF = NULL;
		int *initialSearcher = NULL;
		//number range of random numbers needed [0,mems)
		int mems = numsamples;

		float totalTime;
		hipEvent_t begin, end;
		hipEventCreate(&begin);
		hipEventCreate(&end);

		for (int permNum = 0; permNum < perms; permNum++)
		{
			n = 0;
			
			int *randNums = (int *)malloc(sizeof(int) * numsamples);
			for (int c = 0; c < mems; c++) {
				randNums[c] = rand() % mems;
			}
			while (n < mems) {
				int r = rand() % mems;

				for (x = 0; x < n; x++)
				{
					if (randNums[x] == r){
						break;
					}
				}
				if (x == n){
					randNums[n++] = r;
				}
			}

			

			//after first permutation scramble samplings
			if (permNum > 0)
			{
				for (int counter = 0; counter < numclass1; counter++)
				{
					assert(counter < numsamples);
					classids[randNums[counter]] = 0;
				}
				for (int counter = numclass1; counter < numsamples; counter++)
				{
					assert(counter < numsamples);
					classids[randNums[counter]] = 1;
				}
			}
				
			
			
			free(randNums); randNums = NULL;
			
			//sort data into class1 and class 2
			int index = 0;
			for (i = 0; i < genesetlength; i++) {
				if (genesetindexintodata[i] == -1) {
					i++;
				}

				int jindex1 = 0;
				int jindex2 = 0;
				for (j = 0; j < numsamples; j++) {
					/*if (j == 4268 || j = 4 || classids[j] == -858993460)
					{
					printf("bad value in classids accessed @ %d with value of %d\n", i, classids[j]);
					}*/
					if (classids[j] == 0) {
						assert(index * numclass1 + jindex1 < numclass1 * genesetlength);//transferdata
						assert(genesetindexintodata[i] * numsamples + j < numgenes * numsamples); //data
						transferdata1[index*numclass1 + jindex1] = data[genesetindexintodata[i] * numsamples + j];
						jindex1++;
						//			printf("%d ",transferdata1[index*numclass1+jindex1]);
					}
					if (classids[j] == 1) {
						assert(index*numclass2 + jindex2 < numclass2 * genesetlength);//transferdata
						assert(genesetindexintodata[i] * numsamples + j < numgenes * numsamples); //data
						transferdata2[index*numclass2 + jindex2] = data[genesetindexintodata[i] * numsamples + j];
						jindex2++;
					}
				}
				//	printf("\n");
				index++;
			}

			//dead code never run
			//while (fgets(buf, sizeof(buf), fp)) {
			//	token = strtok(buf, "\t");
			//	strcpy(genenames[numgenes], token);
			//	//    printf("%s\n", genenames[numgenes]);
			//	numgenes++;
			//}
			//printf("Data Grab Done\n");
			int genes = genesetlength;
			//end data grab routine ******************************************************************************
			//samples sizes in both classes


			//start timing- Tomas
			hipEventRecord(begin, 0);



			int samples = numclass1;
			int samples2 = numclass2;

			int c = (((genes*genes) - genes) / 2);

			int scaler = (samples + 1);
			int scaler2 = (samples2 + 1);
			int scalerSum = scaler + scaler2;
			
			//int pos = 0;
			//int posrr = 0;
			//transfer data for processing
			//triAry2 = transferdata1;
			//triAry3 = transferdata2;
			//int arypos = 0;
			//int hold = 0;
			//int spacer = 0;
			//int len = genes;
			
			int* spacer1;//dyn
			spacer1 = (int *)malloc(c*sizeof(int));
			int* ff1;//dyn
			int *searcher;
			ff1 = (int *)malloc(c*sizeof(int));
			searcher = (int *)malloc(genes*sizeof(int));


			//following block of code determines diaganol representation of data matrix
			//searcher[0] = 0;
			//int diff = 0;
			//for (spacer = 0; spacer <= len + 1; spacer++)
			//{
			//	hold = len*spacer;
			//	for (int f = spacer + 1; f < len; f++)
			//	{
			//		spacer1[arypos] = f;
			//		ff1[arypos] = spacer;
			//		arypos++;
			//	}
			//	if (spacer > 0 && spacer < len){
			//		//diff = genes - spacer;
			//		diff = spacer;
			//		printf("diff : %d\n", diff);
			//		searcher[spacer] = searcher[spacer - 1] + diff;
			//	}
			//}
			//for (int i = 0; i < genes; i++)
			//{
			//	printf("searcher[%d] : %d\n", i, searcher[i]);
			//}

			//determines diaganol representation of data matrix
			searcher[0] = 0;
			int position = 0;
			for (int row = 1; row < genes; row++)
			{
				for (int col = 0; col < row; col++)
				{
					assert(position < c);
					spacer1[position] = row;
					//printf("spacer1[%d] : %d\n", position, spacer1[position]);
					ff1[position] = col;
					position++;
					
					
				}
				if (row > 0)
				{
					assert(row < genes);
					searcher[row] = searcher[row - 1] + row;
				}
			}
			
			
			
			//if first permutation store spacr, ff, searcher to use in edgeList calcs after permutations
			if (permNum == 0)
			{
				//initialSpacr = (int *)malloc(sizeof(int) * c);
				//initialFF = (int *)malloc(sizeof(int) * c);
				initialSearcher = (int *)malloc(sizeof(int) * genes);
				//memcpy(initialSpacr, spacer1, sizeof(int) * c);
				//memcpy(initialFF, ff1, sizeof(int) * c);
				memcpy(initialSearcher, searcher, sizeof(int) * genes);
			}

			//start cuda time
			hipEvent_t start, stop;
			float time;
			hipEventCreate(&start);
			hipEventCreate(&stop);


			///cuda launch 1***************************************************************************
			//holds edge data in binary format
			//int onesSize = sizeof(double) * c * scalerSum;
			int *edgesPN;
			edgesPN = (int *)malloc(sizeof(int)* (scalerSum + 1));

			//device copies for out23 and edgesPN
			int *dout23;
			int *dedgesPN;

			//device copies
			int *dtriA, *ddofout, *dtriAb, *dppn, *dstf;
			int *dff, *dspacr;
			//double *d_ones;
			int *dpriorMatrix;
			
			//mem sizes required
			int size2 = c*((samples2 + 1) + (samples + 1))*sizeof(int);
			//int size3 = c*((samples2 + 1) + (samples + 1))*sizeof(double);
			int dppnLength = genesetlength * 2;
			////space alloc for device
			HANDLE_ERROR(hipMalloc((void **)&dtriA, genesetlength*samples*sizeof(int)));
			HANDLE_ERROR(hipMalloc((void **)&dtriAb, genesetlength*samples2*sizeof(int)));
			HANDLE_ERROR(hipMalloc((void **)&dppn, genesetlength * 2 * sizeof(int)));
			HANDLE_ERROR(hipMalloc((void **)&dstf, genesetlength * 2 * 3 * sizeof(int)));
			HANDLE_ERROR(hipMalloc((void **)&ddofout, size2));
			HANDLE_ERROR(hipMalloc((void **)&dff, c*sizeof(int)));
			HANDLE_ERROR(hipMalloc((void **)&dspacr, c*sizeof(int)));
			//hipMalloc((void **)&d_ones, onesSize);
			HANDLE_ERROR(hipMalloc((void **)&dout23, sizeof(int) * c * scalerSum));
			HANDLE_ERROR(hipMalloc((void **)&dedgesPN, sizeof(int) * (scalerSum + 1)));

			HANDLE_ERROR(hipMalloc((void **)&dpriorMatrix, sizeof(int) * genesetlength * genesetlength));
			

			//copy into device 
			assert(genes*samples*sizeof(int) == genesetlength * numclass1 * sizeof(int));
			assert(genes*samples2*sizeof(int) == genesetlength * numclass2 * sizeof(int));
			HANDLE_ERROR(hipMemcpy(dtriA, transferdata1, genes*samples*sizeof(int), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dtriAb, transferdata2, genes*samples2*sizeof(int), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dff, ff1, c*sizeof(int), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dspacr, spacer1, c*sizeof(int), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dpriorMatrix, priorMatrix, genesetlength * genesetlength * sizeof(int), hipMemcpyHostToDevice));
			
			

			//no longer used once copied to GPU
			free(spacer1); spacer1 = NULL;
			free(ff1); ff1 = NULL;

			//deploy
			int milliSecondsElapsed1 = getMilliSpan(start1);
			int start2 = getMilliCount();
			int sampleSum = samples + samples2 + 2;
			//printf("samples : %d\n", samples);

			
			
			//run no states in separate kernel to avoid threading
			//noStates_kernel <<<genes * 2, 1 >>>(genes, samples, samples2, dtriA, dtriAb, dppn, dstf);





			hipEventRecord(start, 0);
			//printf("c = %d\n", c);
			if( c < MAX_THREADS)
			{
			run2 << <sampleSum, c, genes * genes * sizeof(int) >> >(genes, samples, samples2, dtriA, dtriAb, dspacr, dff, ddofout, dppn, dstf, dout23, c, dpriorMatrix, pw, thresh);
			}
			else
			{
				int BPN = ceil((c * 1.0) / MAX_THREADS);
				int TPB = ceil((c * 1.0) / BPN);
			
				//printf("launching with %d blocks per network and %d threads per block\n", BPN, TPB);
				run2Scalable <<< sampleSum * BPN, TPB>>>(genes, samples, samples2, dtriA, dtriAb,dspacr, dff, ddofout, dppn, dstf, dout23, c, dpriorMatrix, pw, thresh, BPN, TPB);
				//printf("run2Scalable completed\n");
			}


			//test ppn/stf
			/*int *tempPpn = (int *)malloc(sizeof(int) * 2 * genesetlength);
			int *tempStf = (int *)malloc(sizeof(int) * 2 * 3 * genesetlength);
			hipMemcpy(tempPpn, dppn, sizeof(int) * 2 * genesetlength, hipMemcpyDeviceToHost);
			hipMemcpy(tempStf, dstf, sizeof(int) * 2 * 3 * genesetlength, hipMemcpyDeviceToHost);
			for (int i = 0; i < 2 * genesetlength; i++)
			{
				printf("ppn[%d] : %d\n", i, tempPpn[i]);
			}
			for (int i = 0; i < 2 * 3 * genesetlength; i++)
			{
				printf("stf[%d] : %d\n", i, tempStf[i]);
			}*/

			//printf("run2 finished\n");
			hipError_t errSync = hipGetLastError();
			if (errSync != hipSuccess)
			{
				printf("%s\n", hipGetErrorString(errSync));
			}
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time, start, stop);
			//printf("Run 2 Time : %f\n", time);

			if (permNum == 0)
			{
				//holds post run2 edge data for edge list calculations after permutations
				edgeListData1 = (int *)malloc(sizeof(int) * c);
				edgeListData2 = (int *)malloc(sizeof(int) * c);

				//host array to transfer output of run2 to edgeListData1/edgeListData2
				int *tempOut23 = (int *)malloc(sizeof(int) * c * scalerSum);
				

				//copy binary data back to CPU
				HANDLE_ERROR(hipMemcpy(tempOut23, dout23, sizeof(int) * c * scalerSum, hipMemcpyDeviceToHost));
				

				//first network in first class - no samples left out
				for (int i = 0; i < c; i++)
				{
					//load 1st network from class 1
					edgeListData1[i] = tempOut23[i];
				}
				int count = 0;
				//last network in 2nd class - no samples left out
				for (int i = (scalerSum - 1) * c; i < (scalerSum) * c; i++)
				{
					edgeListData2[count++] = tempOut23[i];
				}
			


				////copy data for the first network in the first class
				//int *ptr1 = &tempOut23[0];
				//memcpy(edgeListData1, ptr1, sizeof(int) * c);
				////copy data for the first network in the second class
				//printf("2nd memcpy starting point : %d\n", scaler * c);
				////int *ptr2 = &tempOut23[scaler * c];
				//int *ptr2 = &tempOut23[(scaler) * c];
				//memcpy(edgeListData2, ptr2, sizeof(int) * c);
				//ptr1 = NULL;
				//ptr2 = NULL;

free(tempOut23); tempOut23 = NULL;
			}



			int milliSecondsElapsed2 = getMilliSpan(start2);
			int start3 = getMilliCount();



			//device copy
			int *dsrchAry, *tempEdgesSums;
			HANDLE_ERROR(hipMalloc((void **)&dsrchAry, genes * sizeof(int)));
			HANDLE_ERROR(hipMemcpy(dsrchAry, searcher, genes * sizeof(int), hipMemcpyHostToDevice));
			tempEdgesSums = (int *)calloc(sampleSum + 1, sizeof(int));

			free(searcher); searcher = NULL;

			hipEvent_t PN_start, PN_stop;
			hipEventCreate(&PN_start);
			hipEventCreate(&PN_stop);
			hipEventRecord(PN_start, 0);
			float PN_time;

			//edgePerNetworkKernel << < sampleSum + 1, c, (c * sizeof(int)) >> >(dout23, dedgesPN, dsrchAry, genes, MAX_PARENTS, c);
			edgePerNetworkKernel << < sampleSum + 1, 1 >> > (dout23, dedgesPN, dsrchAry, genes, MAX_PARENTS, c);
			//printf("edgesPerNetworkKernel finished\n");
			hipEventRecord(PN_stop, 0);
			//HANDLE_ERROR(hipMemcpy(edgesPN, dedgesPN, sizeof(int) * (scalerSum + 1), hipMemcpyDeviceToHost));
			
			//copy edge sums over to CPU to calculate prefix sum for edgesPN	
			HANDLE_ERROR(hipMemcpy(tempEdgesSums, dedgesPN, sizeof(int) * (scalerSum + 1), hipMemcpyDeviceToHost));	
			
			edgesPN[0] = 0;
			for(int i = 1; i < sampleSum + 1; i++)
			{
				edgesPN[i] = edgesPN[i-1] + tempEdgesSums[i-1]; //prefix sum calculation
			}
			//get rid of this temp array	
			free(tempEdgesSums); tempEdgesSums = NULL;
			//edgesPN on the CPU is now fixed but dedgesPN is used later- copy edgesPN results back to GPU memory
			HANDLE_ERROR(hipMemcpy(dedgesPN, edgesPN, sizeof(int) * (sampleSum + 1), hipMemcpyHostToDevice));
			
			/*
			for (int i = 0; i < scalerSum + 1; i++)
			{
			printf("edgesPN[%d] : %d\n", i, edgesPN[i]);
			}	
			*/
			//exit(EXIT_FAILURE);
			
			errSync = hipGetLastError();
			if (errSync != hipSuccess)
			{
				printf("%s\n", hipGetErrorString(errSync));
			}

			/*for (int i = 0; i < scalerSum + 1; i++)
			{
			printf("edgesPN[%d] : %d\n", i, edgesPN[i]);
			}*/
			//hipEventRecord(PN_stop, 0);
			hipEventSynchronize(PN_stop);
			hipEventElapsedTime(&PN_time, PN_start, PN_stop);
			//printf("edgesPerNetworkKernel time : %f\n", PN_time);
			//hipFree(d_ones); 
			HANDLE_ERROR(hipFree(dpriorMatrix)); dpriorMatrix = NULL;
			HANDLE_ERROR(hipFree(ddofout)); ddofout = NULL;
			HANDLE_ERROR(hipFree(dff)); dff = NULL;
			HANDLE_ERROR(hipFree(dspacr)); dspacr = NULL;//hipFree(dtriA); hipFree(dtriAb);-used later in run4
			/***********************************************************************************************************************************************************/
			//total number of edges
			int numEdges = edgesPN[scalerSum];

			//int N = c;
			//int M = genesetlength - 1;
			//int size1 = sizeof(int)*N*(scalerSum);
			//int size222 = sizeof(double)*N*(scalerSum);
			//*****************************************************************************************
			//run22 launch- create parent graphs
			int noNodes = genesetlength;
			//host copies
			int *pNodes, *pEdges;


			//dev copies
			int *dpEdges, *dpNodes;

			//mem reqs
			int nodeSize = sizeof(int)*(noNodes*(scalerSum));
			int edgeSize = sizeof(int)*numEdges;


			//space alloc for device
			HANDLE_ERROR(hipMalloc((void **)&dpEdges, edgeSize));
			HANDLE_ERROR(hipMalloc((void **)&dpNodes, nodeSize));


			//space alloc for host
			pNodes = (int *)malloc(nodeSize);
			pEdges = (int *)malloc(edgeSize);
			//FILE *edgePNFile = fopen("edgePN2.txt", "w");
			//for(int i = 0; i < scalerSum + 1; i++)
			//{
			//	fprintf(edgePNFile, "edgesPN[%d] : %d\n", i, edgesPN[i]);
			//}
			//fclose(edgePNFile);
			run22 << <scalerSum, noNodes >> >(c, dedgesPN, dout23, dpNodes, noNodes, numEdges, dsrchAry, dpEdges, MAX_PARENTS);
			//printf("run22 finished\n");

			HANDLE_ERROR(hipMemcpy(pNodes, dpNodes, nodeSize, hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(pEdges, dpEdges, edgeSize, hipMemcpyDeviceToHost));

			/*for (int i = 0; i < nodeSize / sizeof(int); i++)
			{
				if (i > edgeSize / sizeof(int))
				{
					printf("nodes[%d] : %d\n", i, pNodes[i]);
				}
				else
				{
					printf("nodes[%d] : %d edges[%d] : %d\n", i, pNodes[i], i, pEdges[i]);
				}
			}*/

			/*if (permNum == 0)
			{
				
				for (int i = 0; i < noNodes; i++)
				{
					printf("pNodes[%d] : %d\n", i, pNodes[i]);
				}
				for (int i = 11 * noNodes; i < (11 * noNodes) + noNodes; i++)
				{
					printf("pNodes[%d] : %d\n", i, pNodes[i]);
				}
			}*/
			

			//ensure parent limit
			checkParentLimit(scalerSum, noNodes, MAX_PARENTS, pNodes, nodeSize / sizeof(int));
			/*for (int i = 0; i < nodeSize / sizeof(int); i++)
			{
				if (i > edgeSize / sizeof(int))
				{
					printf("pNodes[%d] : %d\n", i, pNodes[i]);
				}
				else
				{
					printf("pNodes[%d] : %d\t pEdges[%d] : %d\n", i, pNodes[i], i, pEdges[i]);
				}
			}*/
			/*FILE *outputFile = fopen("NodesEdges2.txt", "w");
			for(int i = 0; i < nodeSize / sizeof(int); i++)
			{
				fprintf(outputFile, "pNodes[%d] : %d\n", i, pNodes[i]);
			}
			for(int i = 0; i < edgeSize / sizeof(int); i++)
			{
				fprintf(outputFile, "pEdges[%d] : %d\n", i, pEdges[i]);
			}

			fclose(outputFile); */
			//printf("%d\n", numEdges);
			
			HANDLE_ERROR(hipFree(dsrchAry)); dsrchAry = NULL;
			HANDLE_ERROR(hipFree(dout23)); dout23 = NULL;
			//end run 22**********************************************************************************************/

			//start processs to identify unique networks
			int scalerCombo = (scalerSum*scalerSum - scalerSum) / 2;
			//host
			int *scalerTest; //compare value
			int *shrunk;
			int *shrunkPlc; //compare to
			scalerTest = (int *)malloc(sizeof(int)*scalerCombo);
			shrunk = (int *)malloc(sizeof(int)*scalerCombo);
			shrunkPlc = (int *)malloc(sizeof(int)*scalerCombo);


			//see line 132 for more info
			idPrep(scalerSum, scalerCombo, scalerTest, shrunkPlc);

			//dev copies
			//launch for run 25 *****************************************************************************************
			int *dshrunk;
			int *dscalerTest;
			int *dshnkplc;
			HANDLE_ERROR(hipMalloc((void**)&dshrunk, sizeof(int)*scalerCombo));
			HANDLE_ERROR(hipMalloc((void**)&dscalerTest, sizeof(int)*scalerCombo));
			HANDLE_ERROR(hipMalloc((void**)&dshnkplc, sizeof(int)*scalerCombo));

			//cp into device
			HANDLE_ERROR(hipMemcpy(dscalerTest, scalerTest, sizeof(int)*scalerCombo, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dshnkplc, shrunkPlc, sizeof(int)*scalerCombo, hipMemcpyHostToDevice));
			//************************************************************************************************
			//(int scaler,int noEdges, int gLength,int scalerCombo, int *dedgesPN, int  *dNodes, int *dedgeAry, int *shrunk)
			// 

			//printf("%/ max: %d   ", maxBlocks*maxThreads);
			//printf("\n");
			run25 << <(scalerCombo / (maxThreads - 1)) + 1, maxThreads - 1 >> >(samples + 1, scalerSum, numEdges, genesetlength, scalerCombo, dedgesPN, dpNodes, dpEdges, dshrunk, dscalerTest, dshnkplc);
			//printf("run25 finished\n");
			//*********************************************************************************************
			HANDLE_ERROR(hipMemcpy(shrunk, dshrunk, sizeof(int)*scalerCombo, hipMemcpyDeviceToHost));
			//*************************test****************************************
			hipFree(dshrunk); dshrunk = NULL;
			hipFree(dscalerTest); dscalerTest = NULL;
			hipFree(dshnkplc); dshnkplc = NULL;
			hipFree(dedgesPN); dedgesPN = NULL;
			hipFree(dpEdges); dpEdges = NULL;
			hipFree(dpNodes); dpNodes = NULL;
			free(shrunkPlc); shrunkPlc = NULL;



			bool *uniqueN, *visted;

			//routine for creatation of unique structures 
			uniqueN = (bool *)malloc(sizeof(bool)*scalerSum);

			uniqueN[0] = true;
			visted = (bool *)malloc(sizeof(bool)*scalerSum);
			visted[0] = true;

			
			for (int p = 0; p < scalerSum; p++){
				visted[p] = false;
			}
			for (int p = 0; p < scalerCombo; p++){
				assert(scalerTest[p] < scalerSum);
				if (visted[scalerTest[p]] == true){
					continue;
				}
				else
				{
					if (shrunk[p] == 0){
						uniqueN[scalerTest[p]] = false;
						visted[scalerTest[p]] = true;
					}
					else
					{
						uniqueN[scalerTest[p]] = true;
					}
				}


			}
			//grab network ids from 1st permutation before unique graphs are identified- used when network file is written
			if (permNum == 0)
			{
				uniqueNetIds = (int *)malloc(sizeof(int) * scalerSum);
				int counter = 0;
				for (int i = 0; i < scalerSum; i++)
				{
					if (uniqueN[i])
					{
						uniqueNetIds[counter++] = i;
					}
				}
				uniqueNetIds = (int *)realloc(uniqueNetIds, counter * sizeof(int));
			}


			free(scalerTest); scalerTest = NULL;
			free(shrunk); shrunk = NULL;
			free(visted); visted = NULL;

			int unisum = 0;
			int edSum = 0;
			//should it be scalerSum or scalerSum + 1?
			for (int p = 0; p < scalerSum; p++){

				if (uniqueN[p] == 1){
					unisum++;
					if (p == scalerSum - 1){
						assert(p < scalerSum + 1);
						edSum = edSum + (numEdges - edgesPN[p]);

					}
					else
					{
						assert(p < scalerSum + 1);
						edSum = edSum + edgesPN[p + 1] - edgesPN[p];

					}
				}
			}

			if (permNum == 0)
			{
				printf("Original Number of unique Networks : %d\n", unisum);
			}
			
			//printf("Number of unique networks : %d\n edSum : %d\n numEdges : %d\n edgesPN : %d\n", unisum, edSum, numEdges, edgesPN[scalerSum]);
			//**********************************restructure all **************************************************
			//printf("edSum %d numEdges %d\n", edSum, numEdges);
			int *pUniNodes, *pUniEdges, *pUniEpn;
			//space alloc
			pUniNodes = (int *)malloc(sizeof(int)*unisum*noNodes);
			pUniEdges = (int *)malloc(sizeof(int)*edSum);
			int uniEpnSize = unisum + 1;
			//pUniEpn = (int *)malloc(sizeof(int)*unisum + 1);
			pUniEpn = (int *)malloc(sizeof(int) * uniEpnSize);
			//printf("size of pUniEpn : %d\n", uniEpnSize);


			structureUnique(unisum, numEdges, scaler, scalerSum, noNodes, uniqueN, edgesPN, pEdges, pNodes, pUniEdges, pUniNodes, pUniEpn);
			//printf("structureUnique (NOT A KERNEL) finished\n");
			/*for (int i = 0; i < uniEpnSize; i++)
			{
				printf("pUniEpn[%d] : %d\n", i, pUniEpn[i]);
			}*/
			free(edgesPN); edgesPN = NULL;
			free(pNodes); pNodes = NULL;
			free(pEdges); pEdges = NULL;
			free(uniqueN); uniqueN = NULL;

			//ensure parent limit
			checkParentLimit(unisum, noNodes, MAX_PARENTS, pUniNodes, unisum * noNodes);
			/*for (int i = 0; i < unisum * noNodes; i++)
			{
				if (i > edSum)
				{
					printf("pNodes[%d] : %d\n", i, pUniNodes[i]);
				}
				else
				{
					printf("pNodes[%d] : %d\t pEdges[%d] : %d\n", i, pUniNodes[i], i, pUniEdges[i]);
				}
			}

			for (int i = 0; i < edSum; i++)
			{
				printf("pUniEdges[%d] : %d\n", i, pUniEdges[i]);
			}
			printf("%d\n", edSum);*/




			if (permNum == 0)
			{
				//store graph data for network file write after permutations finished
				//first_uniEpn = (int *)malloc(sizeof(int) * unisum);
				first_uniEpn = (int *)malloc(sizeof(int) * uniEpnSize);
				first_uniNodes = (int *)malloc(sizeof(int) * unisum * noNodes);
				first_uniEdges = (int *)malloc(sizeof(int) * edSum);
				memcpy(first_uniEpn, pUniEpn, unisum * sizeof(int));
				memcpy(first_uniNodes, pUniNodes, unisum * noNodes * sizeof(int));
				memcpy(first_uniEdges, pUniEdges, edSum * sizeof(int));
				first_numEdges = edSum;
				first_unisum = unisum;

			}




			

			scaler = unisum;
			if (permNum == 0)	{ first_scaler = scaler; }
			numEdges = edSum;
			int uniNodeSize = sizeof(int)*(noNodes*unisum);
			int uniEdgeSize = sizeof(int)*numEdges;

			//cuda run 4(final)*************************************************************************************
			double *out5;

			//dev copies
			//int *dtri1; int *dtri2; 
			double *dout5; int *dpEdges2; int *dpNodes2; int *dNij; int *dNijk;
			int *dUniEpn;
			//space alloc dev
			HANDLE_ERROR(hipMalloc((void **)&dpEdges2, uniEdgeSize));
			HANDLE_ERROR(hipMalloc((void **)&dpNodes2, uniNodeSize));
			HANDLE_ERROR(hipMalloc((void **)&dUniEpn, sizeof(int)*unisum));
			//HANDLE_ERROR(hipMalloc((void **)&dUniEpn, sizeof(int)*uniEpnSize));
			HANDLE_ERROR(hipMalloc((void **)&dout5, sizeof(double)*noNodes*scaler * 2));
			HANDLE_ERROR(hipMalloc((void **)&dNij, sizeof(int)*noNodes*scaler * 54));
			HANDLE_ERROR(hipMalloc((void **)&dNijk, sizeof(int)*noNodes *scaler * 162));

			//space alloc host
			out5 = (double *)malloc(sizeof(double)*noNodes*scaler * 2);



			//cp to devp'4
			

			HANDLE_ERROR(hipMemcpy(dpEdges2, pUniEdges, uniEdgeSize, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dpNodes2, pUniNodes, uniNodeSize, hipMemcpyHostToDevice));
			//HANDLE_ERROR(hipMemcpy(dUniEpn, pUniEpn, sizeof(int)*uniEpnSize, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dUniEpn, pUniEpn, sizeof(int) * unisum, hipMemcpyHostToDevice));
			free(pUniNodes); pUniNodes = NULL;
			free(pUniEdges); pUniEdges = NULL;
			free(pUniEpn); pUniEpn = NULL;

			hipEvent_t run4Start, run4End;
			hipEventCreate(&run4Start);
			hipEventCreate(&run4End);
			hipEventRecord(run4Start, 0);
			float run4Time;

			run4 << <scaler * 2, noNodes >> >(scaler, dUniEpn, genesetlength, edSum, unisum, samples, samples2, dtriA, dtriAb, dpEdges2, dpNodes2, dppn, dstf, dNij, dNijk, dout5, dppnLength);
			//printf("run 4 finished\n");
			hipEventRecord(run4End, 0);
			hipEventSynchronize(run4End);
			hipEventElapsedTime(&run4Time, run4Start, run4End);
			//printf("run 4 time : %f\n", run4Time);
			HANDLE_ERROR(hipMemcpy(out5, dout5, sizeof(double)*noNodes*scaler * 2, hipMemcpyDeviceToHost));

			HANDLE_ERROR(hipFree(dNij)); dNij = NULL;
			HANDLE_ERROR(hipFree(dNijk)); dNijk = NULL;
			HANDLE_ERROR(hipFree(dppn)); dppn = NULL;
			HANDLE_ERROR(hipFree(dstf)); dstf = NULL;
			HANDLE_ERROR(hipFree(dout5)); dout5 = NULL;
			HANDLE_ERROR(hipFree(dpEdges2)); dpEdges2 = NULL;
			HANDLE_ERROR(hipFree(dpNodes2)); dpNodes = NULL;
			HANDLE_ERROR(hipFree(dUniEpn)); dUniEpn = NULL;
			HANDLE_ERROR(hipFree(dtriA)); dtriA = NULL;
			HANDLE_ERROR(hipFree(dtriAb)); dtriAb = NULL;
			
			hipError_t last = hipGetLastError();
			if (last != hipSuccess)
			{
				printf("%s\n", hipGetErrorString(last));
			}
			//int div = 0;
			// end final cuda run ***********************************************************************

			// begin divergence calc
			double *lval1;
			lval1 = (double *)malloc(sizeof(double)*scaler * 2);

			for (int i = 0; i < scaler * 2; i++)
			{
				lval1[i] = 0.0;

			}

			// compute likelihood of different dataset parsed by 2 iterations 
			for (int g = 0; g < 2; g++){
				int set = 0;
				int place = 0;
				double scoreSum = 0;
				double *likeli1;
				double min = 0;
				double max = 0;
				double inAlpha = 0;
				double probScale = 0;
				double likeliSum = 0;
				double nonInf = 0;
				double *dist;

				double *adjusted;
				double *infFlag;
				double *outq;
				int localoffset;
				outq = out5;
				if (g < 1){

					localoffset = 0;
				}
				else
				{

					localoffset = scaler;

				}

				dist = (double *)malloc(sizeof(double)*scaler);
				likeli1 = (double *)malloc(sizeof(double)*scaler);
				adjusted = (double *)malloc(sizeof(double)*scaler);
				infFlag = (double *)malloc(sizeof(double)*scaler);
				for (int k2 = 0; k2 < scaler; k2++){

					dist[k2] = 0.0;

					adjusted[k2] = 0.0;
					infFlag[k2] = 0.0;

				}
				for (int i = 0; i < scaler*noNodes; i++){

					dist[place] += outq[i + localoffset*noNodes];
					set++;
					if (set == noNodes){
						set = 0;
						place++;
					}
				}

				min = dist[0];
				max = dist[0];
				for (int j3 = 0; j3 < scaler; j3++){
					//			printf(" dis %d %f \n", j3, dist[j3]);
					scoreSum += dist[j3];
					if (dist[j3]>max){

						max = dist[j3];
					}
					if (dist[j3] < min){

						min = dist[j3];
					}

				}


				inAlpha = -1 * (scoreSum / scaler);
				//printf("\n min-%f max-%f", min, max);
				//printf("inAlpha-%f", inAlpha);
				probScale = (10) / (max - min);

				for (int m = 0; m < scaler; m++){

					adjusted[m] = (dist[m] + inAlpha)*probScale;
					//			printf("\n adjusted: %f", adjusted[m]);
					likeli1[m] = exp(adjusted[m]);
					//			printf("\n likeli: %f", likeli1[m]);
					nonInf += likeli1[m];
					//likeLi[m] = posInf;
					//suppress overflow infinity error
					#pragma warning(suppress: 4056)
					if (likeli1[m] >= INFINITY || likeli1[m] <= -INFINITY){
						infFlag[m] = 1.0;
						likeliSum++;
					}

				}
				free(dist); dist = NULL;
				free(adjusted); adjusted = NULL;
				//	printf("\n likesum: %f nonInf: %f", likeliSum, nonInf);

				if (likeliSum == 0){
					for (int meow = 0; meow < scaler; meow++){
						likeli1[meow] = likeli1[meow] / nonInf;
						lval1[meow + localoffset] = likeli1[meow];

					}

				}
				else{
					for (int meow = 0; meow < scaler; meow++){
						likeli1[meow] = infFlag[meow] / likeliSum;
						lval1[meow + localoffset] = likeli1[meow];
					}
				}

				free(likeli1); likeli1 = NULL;
				free(infFlag); infFlag = NULL;
				outq = NULL;
			}

			if (permNum == 0)
			{
				first_lval1 = (double *)malloc(sizeof(double) * scaler * 2);
				memcpy(first_lval1, lval1, sizeof(double) * scaler * 2);
			}


			/*******************************************************************************************************************************************/
			double *sea;
			sea = (double*)malloc(sizeof(double)*scaler);

			//scaler unique number of networks
			for (int i = 0; i < scaler; i++)
			{
				sea[i] = (lval1[i] + lval1[i + scaler]) / 2;

			}

			double js = 0;
			double logger = log(2.0);

			//final score
			js = kool(lval1, sea, 0, scaler) / 2 + kool(lval1, sea, scaler, scaler) / 2;
			//printf("\njs: %f\n", js / logger);
			assert(permNum < perms);
			jsVals[permNum] = js / logger;

			//printf("permutation : %d\n", permNum);
			if (isnan(jsVals[0]))
			{
				printf("jsVal[0] NAN--Breaking permutation loop\n");
				jsVals[0] = -999.0;
				break;
			}
			
			
			

			/****************************time**********************************/
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);

			hipEventElapsedTime(&time, start, stop);



			/*************************************************/



			//printf("\n Time1 for the kernel: %f ms\n", time);
			//printf("\n\n");

			free(sea); sea = NULL;
			free(out5); out5 = NULL;
			free(lval1); lval1 = NULL;

			if (permNum % 100 == 0)
			{
				//print every 100 permutations
				printf("Permutation %d finished\n", permNum);
			}

		}//---------------------------------------------------------------------------
		//for loop ends for permutations.
		printf("permutation loop finished\n");
		//free prior knowledge matrix- no longer needed
		
		int nanFlag = 0;
		for (int i = 0; i < perms; i++)
		{
			if (isnan(jsVals[i]))
			{
				nanFlag = 1;
				break;
			}
		}
		if (jsVals[0] < 0 || nanFlag)
		{
			//fprintf(results, "%s %s\n", pathwayName, "GARBAGE VALUE-NAN");
			fprintf(results, "%s failed with %d genes- one of the js scores was nan\n", pathwayName, genesetlength);
			printf("nan value- quiting!\n");
			continue;
		}
		printf("\n\n\n\npermutations finished\n");
		//count how many js values are larger than initial run
		printf("Original JS : %f\n", jsVals[0]);
		/*int largerTally = 0;
		for (int i = 1; i < perms; i++)
		{

			if (jsVals[i] > jsVals[0])
			{
				largerTally++;
			}
		}*/

		//double p_val = 0.0;
		//printf("number larger : %d\n", largerTally);
		//p_val = largerTally / (perms * 1.0);
		//printf("p = %f\n", p_val);
		double p = 0;
		
		if (perms > 0)
		{
			double mu = mean(jsVals, perms);
			double var = variance(mu, jsVals, perms);
			double alpha = (((1 - mu) / var) - (1 / mu)) * pow(mu, 2);
			double beta = alpha * (1 / mu - 1);
			printf("alpha : %f beta : %f\n", alpha, beta);
			//p = 1 - betaCDF(jsVals[0], alpha, beta);
			p = boost::math::ibetac(alpha, beta, jsVals[0]);

			printf("p = %f\n", p);
			fprintf(results, "%s %f %f %d\n", pathwayName, jsVals[0], p, genesetlength);
		}
		
		if (p < pThreshold) //statistically significant
		{
			
			char networkFilePath[600];
			strcpy(networkFilePath, pathwayName);
			strcat(networkFilePath, "_Networks.txt");
			char bdeuFilePath[600];
			strcpy(bdeuFilePath, pathwayName);
			strcat(bdeuFilePath, "_BDEU_SCORES.txt");

			writeNetworkFile(networkFilePath, inputFile, classFile, pathwayName, first_unisum, first_uniEpn, genesetgenes, genesetlength, first_uniNodes, first_uniEdges, first_numEdges, uniqueNetIds);
			writeBdeuScores(bdeuFilePath, inputFile, classFile, pathwayName, class1, class2, first_scaler, first_lval1);

			//printf("\nOriginal JS score : %f\n", jsVals[0]);
			//printf("Original number of unique networks : %d\n", first_unisum);

			//-----------------------------------------------------
			//edgeList calcs
			//edgesPerNetworkKernel --> run22 --> output Edge list
			//NOTE: genesetlength is used to represent the number of nodes aka noNodes as used in prior kernel calls
			//printf("Final run\n");

			//2 networks are being looked at and c = number of different gene combinations
			int c = (((genesetlength*genesetlength) - genesetlength) / 2);
			int numNetworks = 2;

			//host copies
			int *nodes, *edges, *out23;
			int edgesPN[3];

			out23 = (int *)malloc(sizeof(int) * c * numNetworks);

			//dev copies
			int *dout23, *dsrchAry, *dEdgesPN;

			//copy data taken from first run2 permutation
			int *ptr1 = &out23[0];
			int *ptr2 = &out23[c];
			memcpy(ptr1, edgeListData1, sizeof(int) * c);
			memcpy(ptr2, edgeListData2, sizeof(int) * c);
			ptr1 = NULL;
			ptr2 = NULL;

			//allocate device memory and copy
			HANDLE_ERROR(hipMalloc((void **)&dout23, sizeof(int) * c * numNetworks));
			HANDLE_ERROR(hipMalloc((void **)&dEdgesPN, sizeof(int) * (numNetworks + 1)));
			HANDLE_ERROR(hipMalloc((void **)&dsrchAry, genesetlength * sizeof(int)));

			HANDLE_ERROR(hipMemcpy(dout23, out23, sizeof(int) * c * numNetworks, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(dsrchAry, initialSearcher, genesetlength * sizeof(int), hipMemcpyHostToDevice));

			const int PARENTS_LIMIT = INT_MAX;
			
			//edgePerNetworkKernel << <numNetworks + 1, c, (c * sizeof(int)) >> >(dout23, dEdgesPN, dsrchAry, genesetlength, PARENTS_LIMIT, c);
			edgePerNetworkKernel << <numNetworks + 1, 1 >> >(dout23, dEdgesPN, dsrchAry, genesetlength, PARENTS_LIMIT, c);
			//edgePerNetworkKernel << <numNetworks + 1, c, (c * sizeof(int)) + (genesetlength * sizeof(int)) >> >(dout23, dEdgesPN, dsrchAry, genesetlength, PARENTS_LIMIT, c);
			
			//edgePerNetworkKernel calculates sum of edges for each network - now we need to perform the prefix calc for edgesPN on the CPU
			int tempEdgeSum[numNetworks + 1];
			HANDLE_ERROR(hipMemcpy(tempEdgeSum, dEdgesPN, sizeof(int) * (numNetworks + 1), hipMemcpyDeviceToHost));
			edgesPN[0] = 0;
			//calc prefix sum
			for(int i = 1; i < numNetworks + 1; i++){
				edgesPN[i] = edgesPN[i-1] + tempEdgeSum[i-1];
			}//copy results of prefix sum back to GPU for use in run22
			HANDLE_ERROR(hipMemcpy(dEdgesPN, edgesPN, sizeof(int) * (numNetworks + 1), hipMemcpyHostToDevice));


			/*for (int i = 0; i < 3; i++)
			{
				printf("edgesPN[%d]  : %d\n", i, edgesPN[i]);
			}
			
			for (int i = 0; i < genesetlength; i++)
			{
				printf("%d : %s\n", i, genesetgenes[i]);
			}*/



			//needed to calculate how long to make edge array
			int totalEdges = edgesPN[2];

			nodes = (int *)malloc(sizeof(int) * genesetlength * 2);
			edges = (int *)malloc(sizeof(int) * totalEdges);


			int *dNodes, *dEdges;
			HANDLE_ERROR(hipMalloc((void **)&dNodes, sizeof(int) * genesetlength * 2));
			HANDLE_ERROR(hipMalloc((void **)&dEdges, sizeof(int) * totalEdges));

			run22 << <numNetworks, genesetlength >> >(c, dEdgesPN, dout23, dNodes, genesetlength, totalEdges, dsrchAry, dEdges, PARENTS_LIMIT);

			HANDLE_ERROR(hipMemcpy(nodes, dNodes, sizeof(int) * 2 * genesetlength, hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(edges, dEdges, sizeof(int) * totalEdges, hipMemcpyDeviceToHost));


			HANDLE_ERROR(hipFree(dout23)); dout23 = NULL;
			HANDLE_ERROR(hipFree(dsrchAry)); dsrchAry = NULL;
			HANDLE_ERROR(hipFree(dEdgesPN)); dEdgesPN = NULL;
			HANDLE_ERROR(hipFree(dNodes)); dNodes = NULL;
			HANDLE_ERROR(hipFree(dEdges)); dEdges = NULL;
			free(out23); out23 = NULL;

			char edgeListFile[600];
			strcpy(edgeListFile, pathwayName);
			strcat(edgeListFile, "_EdgeList.txt");
			//int networkIds[2] = { 1, 2 };
			//writeNetworkFile(edgeListFile, inputFile, classFile, pathwayName, 2, edgesPN, genesetgenes, genesetlength, nodes, edges, totalEdges, networkIds);
			writeEdgeListFile(edgeListFile, inputFile, classFile, pathwayName, genesetgenes, genesetlength, nodes, edges, edgesPN, priorMatrix, class1, class2);
			
			hipEventRecord(end, 0);
			hipEventSynchronize(end);
			hipEventElapsedTime(&totalTime, begin, end);

			//printf("Total Run Time : %f\n", totalTime);

			//FILE *timeFile = fopen("Time.txt", "w");
			//fprintf(timeFile, "%f", totalTime);
			//output number of unique networks to check against java scores
			printf("Total run time : %f\n", totalTime);
			//fprintf(timeFile, "%f", totalTime);
			//fclose(timeFile);
			free(nodes); nodes = NULL;
			free(edges); edges = NULL;
		}

		printf("\nPathway finished.\n\n");
		free(priorMatrix); priorMatrix = NULL; //free this after writing files b/c needed for writeEdgeListFile

		diff = clock() - cpuTime;
		int msec = diff * 1000 / CLOCKS_PER_SEC;
		printf("Time taken %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
		//---------------------------------------------------------------------------
		//free variables

		

		free(transferdata1);
		free(transferdata2);
		transferdata1 = NULL;
		transferdata2 = NULL;

		free(first_lval1);
		free(first_uniNodes);
		free(first_uniEdges);
		free(first_uniEpn);
		free(jsVals);
		free(uniqueNetIds);
		free(edgeListData1);
		free(edgeListData2);
		//free(initialFF);
		free(initialSearcher);
		//free(initialSpacr);
		first_lval1 = NULL;
		first_uniNodes = NULL;
		first_uniEdges = NULL;
		first_uniEpn = NULL;
		jsVals = NULL;
		uniqueNetIds = NULL;
		edgeListData1 = NULL;
		edgeListData2 = NULL;
		//initialFF = NULL;
		initialSearcher = NULL;
		//initialSpacr = NULL;

	}
	fclose(fp3);
	fclose(results);

	free(data);
	data = NULL;
	//use to make sure all data is recorded and visual
	//profiler works
	//hipDeviceReset();

	return 0;
}
